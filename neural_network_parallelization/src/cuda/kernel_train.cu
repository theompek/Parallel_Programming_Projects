#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include <math.h>
#include <pthread.h>

#define lamda 0.8
#define h(x) (1/(1+expf(-b*x))) // activation function
//To b brisketai sto arxeio utils.h

#define Input(x,y) (Input[x+length*y])
#define Output(x,y) (Output[x+outLength*y])

#define W1(x,y) (W1[x+n2*y])
#define W2(x,y) (W2[x+n3*y])
#define W3(x,y) (W3[x+n4*y])

#define d_W1(x,y) (d_W1[x+n2*y])
#define d_W2(x,y) (d_W2[x+n3*y])
#define d_W3(x,y) (d_W3[x+n4*y])

#define epochMax 500 //Megistos ari8mos epoxwn


__global__ void kernel(float *d_W1, float *d_W2, float *d_W3, float *Input, float *Output,int length,int width,int outLength){
float sum;
__shared__ float U_int[n1];
__shared__ float u2Out[n2];
__shared__ float u3Out[n3];
__shared__ float u4Out[n4];
__shared__ float error[n4];
__shared__ float delta3[n4];
__shared__ float delta2[n3];
__shared__ float delta1[n2];

__shared__ float W1[n2*n1];
__shared__ float W2[n3*n2];
__shared__ float W3[n4*n3];

//Αντιγραφή από την Gloabal memory στην Shared Memory
if((threadIdx.y==0)&&(threadIdx.x<n2))
for(int i=0;i<n1;i++)
W1(threadIdx.x,i)=d_W1(threadIdx.x,i);


if((threadIdx.y==1)&&(threadIdx.x<n3))
for(int i=0;i<n2;i++)
W2(threadIdx.x,i)=d_W2(threadIdx.x,i);


if((threadIdx.y==2)&&(threadIdx.x<n4))
for(int i=0;i<n3;i++)
W3(threadIdx.x,i)=d_W3(threadIdx.x,i);


int epoch=0;

while(epoch<epochMax){
	
for(int Id=0;Id<width;Id++)
{
if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1111111111HiddenLayer

	//-----FORWARD PATH-----	 
	for(int i=0;i<n1;i++)
	U_int[i]=Input(i,Id);
	
	sum=0.0;
	for(int i=0;i<n1;i++)
	sum+= W1(threadIdx.x,i)*U_int[i];
		
	u2Out[threadIdx.x]=h(sum); 
}	

__syncthreads(); //1
	
if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2222222222HiddenLayer

	//-----FORWARD PATH----
	sum=0.0;
	for(int i=0;i<n2;i++)
	sum+= W2(threadIdx.x,i)*u2Out[i];
		
	u3Out[threadIdx.x]=h(sum);
}	
__syncthreads(); //2

if((threadIdx.y==2)&&(threadIdx.x<n4)){//------->33333333Outtt
	//-----FORWARD PATH-----	 
	sum=0.0;
	for(int i=0;i<n3;i++)
	sum+= W3(threadIdx.x,i)*u3Out[i];
		
	u4Out[threadIdx.x]=h(sum); 
	   
	//-----BACKWARD PATH------- 
	error[threadIdx.x]=Output(threadIdx.x,Id)-u4Out[threadIdx.x];
	delta3[threadIdx.x]=error[threadIdx.x]*(b*u4Out[threadIdx.x]*(1.0-u4Out[threadIdx.x])); //u4Out=h(Σ(U3)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))
	
	//printf("delt3-->33333= %f \n----\n",delta3[threadIdx.x]);
}
__syncthreads(); //3

if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2222222222HiddenLayer

//-----BACKWARD PATH-------   
	sum=0.0;
	for(int j=0;j<n4;j++)
	sum+=W3(j,threadIdx.x)*delta3[j];

//printf("delt3-->2222= %f \n----\n",delta3[j]);
	delta2[threadIdx.x]=(b*u3Out[threadIdx.x]*(1.0-u3Out[threadIdx.x]))*sum; //u3Out=h(Σ(U2)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))	

}

__syncthreads(); //4

if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1111111111InputLayer
	//-----BACKWARD PATH-------	
	sum=0.0;
	for(int j=0;j<n3;j++)
	sum+=W2(j,threadIdx.x)*delta2[j];

	
	delta1[threadIdx.x]=(b*u2Out[threadIdx.x]*(1.0-u2Out[threadIdx.x]))*sum; //u2Out=h(Σ(U1)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))
}
__syncthreads(); //5

if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1111111111HiddenLayer
for(int j=0;j<n1;j++)
W1(threadIdx.x,j)+=lamda*delta1[threadIdx.x]*U_int[j];
}

if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2222222222HiddenLayer
for(int j=0;j<n2;j++)
W2(threadIdx.x,j)+=lamda*delta2[threadIdx.x]*u2Out[j];	
}	

if((threadIdx.y==2)&&(threadIdx.x<n4)){//------->33333333HOuttt	
for(int j=0;j<n3;j++)
W3(threadIdx.x,j)+=lamda*delta3[threadIdx.x]*u3Out[j];
}	

} 

epoch++;	
}//Τελος while	


//Αντιγραφή πίσω στην Gloabal memory
if((threadIdx.y==0)&&(threadIdx.x<n2))
for(int i=0;i<n1;i++)
d_W1(threadIdx.x,i)=W1(threadIdx.x,i);


if((threadIdx.y==1)&&(threadIdx.x<n3))
for(int i=0;i<n2;i++)
d_W2(threadIdx.x,i)=W2(threadIdx.x,i);


if((threadIdx.y==2)&&(threadIdx.x<n4))
for(int i=0;i<n3;i++)
d_W3(threadIdx.x,i)=W3(threadIdx.x,i);
		
}//Telos kernel
