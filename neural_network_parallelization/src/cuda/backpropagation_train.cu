#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include <math.h>
#include <pthread.h>

#define lamda 0.8
#define h(x) (1/(1+expf(-b*x))) // activation function
//To b brisketai sto arxeio utils.h

#define Input(x,y) (Input[x+length*y])
#define Output(x,y) (Output[x+outLength*y])

#define Input(x,y) (Input[x+length*y])
#define Output(x,y) (Output[x+outLength*y])

#define W1(x,y) (W1[x+n2*y])
#define W2(x,y) (W2[x+n3*y])
#define W3(x,y) (W3[x+n4*y])

#define d_W1(x,y) (d_W1[x+n2*y])
#define d_W2(x,y) (d_W2[x+n3*y])
#define d_W3(x,y) (d_W3[x+n4*y])

#define epochMax 500 //Megistos ari8mos epoxwn

__global__ void kernel(float *d_W1, float *d_W2, float *d_W3, float *Input, float *Output,int length,int width,int outLength){

//θα χρησιμοποιήσουμε την Shared memory για πιο γρήγορη προσπέλαση
float sum;
__shared__ float U_int[n1];
__shared__ float u2Out[n2];
__shared__ float u3Out[n3];
__shared__ float u4Out[n4];
__shared__ float error[n4];
__shared__ float delta3[n4];
__shared__ float delta2[n3];
__shared__ float delta1[n2];

__shared__ float W1[n2*n1];
__shared__ float W2[n3*n2];
__shared__ float W3[n4*n3];

int idx = threadIdx.x+blockDim.x*blockIdx.x;
int ml=blockDim.x*blockDim.y;

//Αρχικοποιουμε με μικρες τιμες τους πινακες
if((threadIdx.y==0)&&(threadIdx.x<n2))
for(int i=0;i<n1;i++)
W1(threadIdx.x,i)=0.01*idx/(ml*(i+2));


if((threadIdx.y==1)&&(threadIdx.x<n3))
for(int i=0;i<n2;i++)
W2(threadIdx.x,i)=0.01*idx/(ml*(i+2));


if((threadIdx.y==2)&&(threadIdx.x<n4))
for(int i=0;i<n3;i++)
W3(threadIdx.x,i)=0.01*idx/(ml*(i+2));

int epoch=0;

while(epoch<epochMax){
	
for(int Id=0;Id<width;Id++)
{
if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1HiddenLayer

	//-----FORWARD PATH-----	 
	for(int i=0;i<n1;i++)
	U_int[i]=Input(i,Id);
	
	sum=0.0;
	for(int i=0;i<n1;i++)
	sum+= W1(threadIdx.x,i)*U_int[i];
		
	u2Out[threadIdx.x]=h(sum); 
}	

__syncthreads(); //1
	
if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2HiddenLayer

	//-----FORWARD PATH----
	sum=0.0;
	for(int i=0;i<n2;i++)
	sum+= W2(threadIdx.x,i)*u2Out[i];
		
	u3Out[threadIdx.x]=h(sum);
}	
__syncthreads(); //2

if((threadIdx.y==2)&&(threadIdx.x<n4)){//------->3Outtt
	//-----FORWARD PATH-----	 
	sum=0.0;
	for(int i=0;i<n3;i++)
	sum+= W3(threadIdx.x,i)*u3Out[i];
		
	u4Out[threadIdx.x]=h(sum); 
	   
	//-----BACKWARD PATH------- 
	error[threadIdx.x]=Output(threadIdx.x,Id)-u4Out[threadIdx.x];
	delta3[threadIdx.x]=error[threadIdx.x]*(b*u4Out[threadIdx.x]*(1.0-u4Out[threadIdx.x])); //u4Out=h(Σ(U3)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))
	
}
__syncthreads(); //3

if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2HiddenLayer

//-----BACKWARD PATH-------   
	sum=0.0;
	for(int j=0;j<n4;j++)
	sum+=W3(j,threadIdx.x)*delta3[j];

//printf("delt3-->2222= %f \n----\n",delta3[j]);
	delta2[threadIdx.x]=(b*u3Out[threadIdx.x]*(1.0-u3Out[threadIdx.x]))*sum; //u3Out=h(Σ(U2)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))	

}

__syncthreads(); //4

if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1HiddenLayer
	//-----BACKWARD PATH-------	
	sum=0.0;
	for(int j=0;j<n3;j++)
	sum+=W2(j,threadIdx.x)*delta2[j];

	
	delta1[threadIdx.x]=(b*u2Out[threadIdx.x]*(1.0-u2Out[threadIdx.x]))*sum; //u2Out=h(Σ(U1)) αρα απο τον τύπο dh(x)=(b*h(x)*(1-h(x)))
}
__syncthreads(); //5

if((threadIdx.y==0)&&(threadIdx.x<n2)){//------->1HiddenLayer
for(int j=0;j<n1;j++)
W1(threadIdx.x,j)+=lamda*delta1[threadIdx.x]*U_int[j];
}

if((threadIdx.y==1)&&(threadIdx.x<n3)){//------->2HiddenLayer
for(int j=0;j<n2;j++)
W2(threadIdx.x,j)+=lamda*delta2[threadIdx.x]*u2Out[j];	
}	

if((threadIdx.y==2)&&(threadIdx.x<n4)){//------->3HOuttt	
for(int j=0;j<n3;j++)
W3(threadIdx.x,j)+=lamda*delta3[threadIdx.x]*u3Out[j];
}	

} 

epoch++;	
}//Τελος while	


//Αντιγραφή πίσω στην Gloabal memory
if((threadIdx.y==0)&&(threadIdx.x<n2))
for(int i=0;i<n1;i++)
d_W1(threadIdx.x,i)=W1(threadIdx.x,i);


if((threadIdx.y==1)&&(threadIdx.x<n3))
for(int i=0;i<n2;i++)
d_W2(threadIdx.x,i)=W2(threadIdx.x,i);


if((threadIdx.y==2)&&(threadIdx.x<n4))
for(int i=0;i<n3;i++)
d_W3(threadIdx.x,i)=W3(threadIdx.x,i);
		
}//Telos kernel



extern "C" void * backpropagation_train(void *dataTh){

PthreadData *DataTrd = (PthreadData* )dataTh;
	
int Max;
if(n2<=n3){
Max=n3;	
}else{Max=n2;}	

int numBlocks = 1;  //Αριθμός block
dim3 threadsPerBlock(Max, 3); //Διαστάσεις block

float *d_W1;
float *d_W2;
float *d_W3;
float *d_Uin;
float *d_Uout;
int length;
int width;
int outLength;
float *Input;
float *Output;
float *W1;
float *W2;
float *W3;


length=DataTrd->length/numOfSubNetworks;
width=DataTrd->width;
outLength=DataTrd->outLength/numOfSubNetworks;

Input=(float *)malloc(length*width*sizeof(float));
Output=(float *)malloc(outLength*width*sizeof(float));

//Αντιγράφουμε τα αντιστοιχα δεδομένα εισόδων-εξόδων
for(int j=0;j<width;j++)
for(int i=0;i<length;i++)
Input(i,j)=DataTrd->Input[(i+length*DataTrd->Id_x)+DataTrd->length*j];	

for(int j=0;j<width;j++)
for(int i=0;i<outLength;i++)
Output(i,j)=DataTrd->Output[(i+outLength*DataTrd->Id_x)+DataTrd->outLength*j];	

W1 = DataTrd->W1;
W2 = DataTrd->W2;
W3 = DataTrd->W3;

hipMalloc(&d_W1, n2*n1*sizeof(float));
hipMalloc(&d_W2, n3*n2*sizeof(float));
hipMalloc(&d_W3, n4*n3*sizeof(float));

hipMalloc(&d_Uin, length*width*sizeof(float));
hipMalloc(&d_Uout, outLength*width*sizeof(float));

hipMemcpy(d_Uin, Input, length*width*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_Uout,Output, outLength*width*sizeof(float), hipMemcpyHostToDevice);

kernel<<<numBlocks, threadsPerBlock>>>(d_W1, d_W2, d_W3, d_Uin, d_Uout,length,width,outLength);

hipMemcpy(W1, d_W1, n2*n1*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(W2, d_W2, n3*n2*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(W3, d_W3, n4*n3*sizeof(float), hipMemcpyDeviceToHost);

free(Input);
free(Output);

hipFree(d_W1);
hipFree(d_W2);
hipFree(d_W3);
hipFree(d_Uin);
hipFree(d_Uout);


pthread_exit(NULL);
}



