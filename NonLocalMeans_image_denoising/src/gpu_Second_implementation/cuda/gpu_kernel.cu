
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 16

#define MAX(i,j) ( (i)<(j) ? (j):(i) )
#define MIN(i,j) ( (i)<(j) ? (i):(j) )

#define SubArrayA(x,y) subArrayA[(x)*BLOCK_SIZE+(y)]
#define SubArrayB(x,y) subArrayB[(x)*BLOCK_SIZE+(y)]


#define InputArrayA(x,y) inputArrayA[(x)*BLOCK_SIZE+(y)]
#define InputArrayB(x,y) inputArrayB[(x)*BLOCK_SIZE+(y)]

#define ImageOut(x,y) imageOut[(x)*imageSize+(y)]
#define ZiArray(x,y) ziArray[(x)*imageSize+(y)]

__device__ float gaussianDistance(float  *inputArrayA,float *inputArrayB,int xj,int yj,int halfPatchWidth);
__device__ float weightingFunct(float  *inputArrayA,float *inputArrayB,int xj,int yj,int halfPatchWidth,float sigma,float Zi);
__device__ float normFactor(float  *inputArrayA,float *inputArrayB,int halfPatchWidth,float sigma);
__device__ float nonLocalMeans(float  *inputArrayA,float *inputArrayB,float *imageOut,int xi,int yi,int halfPatchWidth,int imageSize,float sigma,float Zi);

__global__ void mainGpuFunction(float  const * const inputArray,float *imageOut,float *ziArray,int halfPatchWidth,int imageSize,float sigma,int flag)
{
 //���� ��� ������������� ��� ����������� pixel ��� ������� ���������������
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
    
//������ �� ����� ���� ��� ���� ��� �������	
if((xi<imageSize)&&(yi<imageSize)){

 __shared__ float subArrayA[BLOCK_SIZE*BLOCK_SIZE];
 __shared__ float subArrayB[BLOCK_SIZE*BLOCK_SIZE];

//�� ������ �� ������� Z=0
//if(blockIdx.z==0) ImageOut(xi,yi)=0; 

//===========================
//������������ ������� �� ��� ��������� blockIdx.z ���� block �� 
//���������� ��� ���������� ���� �� �� ����� block.
//������ � ����� blockIdx.z=neighblockX*blockDim.x+neighblockY

int neighblockX;
int neighblockY;


for(int i=0;i<blockDim.x;i++){
for(int j=0;j<blockDim.y;j++){
	 if(blockIdx.z==(i*blockDim.x+j))
	 {
	  neighblockX=i;
	  neighblockY=j;
	  i=blockDim.x;//��� �� ������ ��� �� loop
	  break;
	 }
 }
}
//===========================

int xj = neighblockX*blockDim.x + threadIdx.x;
int yj = neighblockY* blockDim.y + threadIdx.y;

//�� ���� �� ��������� ��� ���� Zi
if(flag==1)
{
//���������� �� ����� block
//���� thread �������� ������ ��� �������� ���� ������
SubArrayA(threadIdx.x,threadIdx.y)=inputArray[xi*imageSize+yi];
__syncthreads();

//���������� ���� ������ � ��� ��� �� block ������� ������� �� ��� 
//������ ��� blockIdx.z ���� ����� �����������
SubArrayB(threadIdx.x,threadIdx.y)=inputArray[xj*imageSize+yj];	
__syncthreads();

float Zi=normFactor(subArrayA,subArrayB,halfPatchWidth,sigma);


atomicAdd(&ZiArray(xi,yi),Zi);

//����������� �� ������������ ��� �� ���������� ��� �� �i
__syncthreads();  	
		
}
else //�� ���� �� ��������� �� ���������� w(i,j)*f(j)
{
//���������� �� ����� block
//���� thread �������� ������ ��� �������� ���� ������
SubArrayA(threadIdx.x,threadIdx.y)=inputArray[xi*imageSize+yi];
__syncthreads();

//���������� ���� ������ � ��� ��� �� block ������� ������� �� ��� 
//������ ��� blockIdx.z ���� ����� �����������
SubArrayB(threadIdx.x,threadIdx.y)=inputArray[xj*imageSize+yj];	
__syncthreads();

float SumWeight=nonLocalMeans(subArrayA,subArrayB,imageOut,xi,yi,halfPatchWidth,imageSize,sigma,ZiArray(xi,yi));


atomicAdd(&ImageOut(xi,yi),SumWeight);

 __syncthreads();  
 
	
}

}


}

__device__ float nonLocalMeans(float  *inputArrayA,float *inputArrayB,float *imageOut,int xi,int yi,int halfPatchWidth,int imageSize,float sigma,float Zi){

 
	 float ww=0;
	

	 for(int xj=0;xj<BLOCK_SIZE;xj++)
     {	
      for(int yj=0;yj<BLOCK_SIZE;yj++)
       {
		  ww+=weightingFunct(inputArrayA,inputArrayB,xj,yj,halfPatchWidth,sigma,Zi)*InputArrayB(xj,yj); //w(i,j)*f(j)
		 
	   }
	 } 
	 
return(ww);
	 	
}

//� ��������� w(i,j)=w([xi,yi] [xj,yj])
__device__ float weightingFunct(float  *inputArrayA,float *inputArrayB,int xj,int yj,int halfPatchWidth,float sigma,float Zi){

float distance=gaussianDistance(inputArrayA,inputArrayB,xj,yj,halfPatchWidth); 	
	
	return ( ( expf(-(distance/(sigma*sigma))) )/Zi);
}

//� ��������� Z(i)=Z(xi,yi)
__device__ float normFactor(float *inputArrayA,float *inputArrayB,int halfPatchWidth,float sigma){

float square_sigma=sigma*sigma;
float z=0;

for(int i=0;i<BLOCK_SIZE;i++)
{
 for(int j=0;j<BLOCK_SIZE;j++)
 {
  float distance=gaussianDistance(inputArrayA,inputArrayB,i,j,halfPatchWidth); 
	 
	z+=expf(-(distance/square_sigma) );   
 }	
}
 
return (z);	
}

//����������� ��� �������� |f(Ni)-f(Nj)|
//�������������� Gaussian Euclidean Distance
__device__ float gaussianDistance(float  *inputArrayA,float *inputArrayB,int xj,int yj,int halfPatchWidth){

int xi=threadIdx.x;
int yi=threadIdx.y;

//� ������� i ���������� ��� pixel ��� ����� ������������ ��� ��� ����
//��� � ������� j ��� �������� pixel �� �� ����� ������� ��������

int ai;
int bi;
int aj;
int bj;

int SumWeight=0;  //�������� �����
float distance=0;//�������� ������� ��������� pixel
float diff=0;     //������� ������ 2 pixel ��������
 
for(int i=-halfPatchWidth;i<=halfPatchWidth;i++)
 {
  for(int j=-halfPatchWidth;j<=halfPatchWidth;j++)
  {
  ai=xi+i; 
  bi=yi+j;
  aj=xj+i;
  bj=yj+j;
  
if((aj<0)||(aj>=BLOCK_SIZE)) aj=xj-i; 
if((bj<0)||(bj>=BLOCK_SIZE)) bj=yj-j;
  
if((ai<0)||(ai>=BLOCK_SIZE)) ai=xi-i; 
if((bi<0)||(bi>=BLOCK_SIZE)) bi=yi-j;  


   if (ai!=xi || bi!=yi)//���� �������� ��� ��� ��� �������� ����� ��� ��������� 
   { 
    int weight=1/(MAX(ai-xi,xi-ai)+MAX(bi-yi,yi-bi));
    SumWeight+=weight;	
	diff=InputArrayA(ai,bi)-InputArrayB(aj,bj);
	distance+=diff*diff*weight;
	
   }
  }
 }


 return (distance/SumWeight);


}

