
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 16

#define MAX(i,j) ( (i)<(j) ? (j):(i) )
#define MIN(i,j) ( (i)<(j) ? (i):(j) )

#define SubArrayA(x,y) subArrayA[(x)*BLOCK_SIZE+(y)]
#define InputArrayA(x,y) inputArrayA[(x)*BLOCK_SIZE+(y)]
#define ImageOut(x,y) imageOut[(x)*imageSize+(y)]


__device__ float gaussianDistance(float  *inputArrayA,int xj,int yj,int halfPatchSWidth);
__device__ float weightingFunct(float  *inputArrayA,int xj,int yj,int halfPatchSWidth,float sigma,float Zi);
__device__ float normFactor(float  *inputArrayA,int halfPatchSWidth,float sigma);
__device__ float nonLocalMeans(float  *inputArrayA,float *imageOut,int halfPatchSWidth,int imageSize,float sigma);

__global__ void mainGpuFunction(float  const * const inputArray,float *imageOut,int halfPatchSWidth,int imageSize,float sigma)
{
 //Πάρε τις συντεταγμένες του αντίστοιχου pixel που κάνουμε αποθορυβοποίηση
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;

//Πρέπει να είμαι μέσα στα όρια της εικόνας    
if((xi<imageSize)&&(yi<imageSize)){


__shared__ float subArrayA[BLOCK_SIZE*BLOCK_SIZE];

//Κάθε ένα απο τα thread θα φορτώσει μία τιμή του πίνακα τιμών των pixel
//απο την global μνήμη στην shared μνημη 	
SubArrayA(threadIdx.x,threadIdx.y)=inputArray[xi*imageSize+yi];
__syncthreads();

//Το κάθε νήμα εφαρμόζει τον αλγόριθμο για το τμήμα της εικόνας που φόρτωσε στην μνήμη shared
ImageOut(xi,yi)=nonLocalMeans(subArrayA,imageOut,halfPatchSWidth,imageSize,sigma);

 }//Τέλος if	
 
}


__device__ float nonLocalMeans(float  *inputArrayA,float *imageOut,int halfPatchSWidth,int imageSize,float sigma){

	 float ww=0;
	 
	 float Zi=normFactor(inputArrayA,halfPatchSWidth,sigma); //Υπολογίζουμε την τιμή Z(i)
     
	 //Τα αθροίσματα w(i,j)*f(j)
	 for(int xj=0;xj<BLOCK_SIZE;xj++)
     {	
      for(int yj=0;yj<BLOCK_SIZE;yj++)
       {
		  ww+=weightingFunct(inputArrayA,xj,yj,halfPatchSWidth,sigma,Zi)*InputArrayA(xj,yj); //w(i,j)*f(j)
		 
	   }
	 } 
	 
return(ww);
	 	 
}

//Η μεταβλητή w(i,j)=w([xi,yi] [xj,yj])
__device__ float weightingFunct(float  *inputArrayA,int xj,int yj,int halfPatchSWidth,float sigma,float Zi){

float distance=gaussianDistance(inputArrayA,xj,yj,halfPatchSWidth); 	
	
	return ( ( exp(-(distance/(sigma*sigma))) )/Zi);
}

//Η μεταβλητη Z(i)=Z(xi,yi)
__device__ float normFactor(float  *inputArrayA,int halfPatchSWidth,float sigma){

float square_sigma=sigma*sigma;
float z=0;

for(int i=0;i<BLOCK_SIZE;i++)
{
 for(int j=0;j<BLOCK_SIZE;j++)
 {
  float distance=gaussianDistance(inputArrayA,i,j,halfPatchSWidth); 
	 
	z+=exp(-(distance/square_sigma) );
	
   
 }	
}

return (z);	
}

//Υπολογισμός της διαφοράς |f(Ni)-f(Nj)|
//Χρησιμοποιούμε Gaussian Euclidean Distance
__device__ float gaussianDistance(float  *inputArrayA,int xj,int yj,int halfPatchSWidth){

int xi=threadIdx.x;
int yi=threadIdx.y;

//Ο δείκτης i αναφέρεται στο pixel του οποίο υπολογίζουμε την νέα τιμή
//ενώ ο δείκτης j στα υπόλοιπα pixel με τα οποία γίνεται σύγκριση

int ai;
int bi;
int aj;
int bj;

int SumWeight=0;  //Άθροισμα βαρών
float distance=0;//Συνολική διαφορά γειτονιάς pixel
float diff=0;     //Διαφορά μεταξύ 2 pixel γειτόνων
 
for(int i=-halfPatchSWidth;i<=halfPatchSWidth;i++)
 {
  for(int j=-halfPatchSWidth;j<=halfPatchSWidth;j++)
  {
  ai=xi+i; 
  bi=yi+j;
  aj=xj+i;
  bj=yj+j;
  
if((aj<0)||(aj>=BLOCK_SIZE)) aj=xj-i; 
if((bj<0)||(bj>=BLOCK_SIZE)) bj=yj-j;
  
if((ai<0)||(ai>=BLOCK_SIZE)) ai=xi-i; 
if((bi<0)||(bi>=BLOCK_SIZE)) bi=yi-j;  

   if (ai!=xi || bi!=yi)//Κάνε σύγκριση για όλη την γειτονιά εκτώς του κεντρικού 
   { 
    int weight=1/(MAX(ai-xi,xi-ai)+MAX(bi-yi,yi-bi));
    SumWeight+=weight;	
	diff=InputArrayA(ai,bi)-InputArrayA(aj,bj);
	distance+=diff*diff*weight;
	
   }
  }
 }

 return (distance/SumWeight);

}

